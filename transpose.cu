#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>
#include <algorithm>
#include <numeric>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <agency/cuda/execution_policy.hpp>

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 100;

// Check errors and print GB/s
void postprocess(const thrust::host_vector<float>& ref, const thrust::host_vector<float>& res, float ms)
{
  auto mismatch = std::mismatch(ref.begin(), ref.end(), res.begin());
  if(mismatch.first != ref.end())
  {
    int i = mismatch.first - ref.begin();
    printf("%d %f %f\n", i, *mismatch.second, *mismatch.first);
    printf("%25s\n", "*** FAILED ***");
  }
  else
  {
    printf("%20.2f\n", 2 * ref.size() * sizeof(float) * 1e-6 * NUM_REPS / ms );
  }
}

// simple copy kernel
// Used as reference case representing best effective bandwidth.
struct copy_kernel
{
  template<class Agent>
  __device__
  void operator()(Agent& self, float* odata, const float* idata)
  {
    auto idx = TILE_DIM * self.outer().index() + self.inner().index();
    int width = self.outer().group_shape()[0] * TILE_DIM;

    for(int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    {
      odata[(idx[1]+j)*width + idx[0]] = idata[(idx[1]+j)*width + idx[0]];
    }
  }
};


// copy kernel using shared memory
// Also used as reference case, demonstrating effect of using shared memory.
__global__ void copySharedMem(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];          
}


struct transpose_naive
{
  template<class Agent>
  __device__
  void operator()(Agent& self, float* odata, const float* idata)
  {
    auto idx = TILE_DIM * self.outer().index() + self.inner().index();
    int width = self.outer().group_shape()[0] * TILE_DIM;

    for(int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      odata[idx[0]*width + (idx[1]+j)] = idata[(idx[1]+j)*width + idx[0]];
  }
};


struct transpose_coalesced
{
  template<class Agent>
  __device__
  void operator()(Agent& self, float* odata, const float* idata)
  {
    __shared__ float tile[TILE_DIM][TILE_DIM];
      
    int x = self.outer().index()[0] * TILE_DIM + self.inner().index()[0];
    int y = self.outer().index()[1] * TILE_DIM + self.inner().index()[1];
    int width = self.outer().group_shape()[0] * TILE_DIM;

    for(int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
      tile[self.inner().index()[1]+j][self.inner().index()[0]] = idata[(y+j)*width + x];
    }

    self.inner().wait();

    x = self.outer().index()[1] * TILE_DIM + self.inner().index()[0];  // transpose block offset
    y = self.outer().index()[0] * TILE_DIM + self.inner().index()[1];

    for(int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
      odata[(y+j)*width + x] = tile[self.inner().index()[0]][self.inner().index()[1] + j];
    }
  }
};


struct transpose_no_bank_conflicts
{
  template<class Agent>
  __device__
  void operator()(Agent& self, float* odata, const float* idata)
  {
    __shared__ float tile[TILE_DIM][TILE_DIM+1];
      
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
       tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

    self.inner().wait();

    x = self.outer().index()[1] * TILE_DIM + self.inner().index()[0];  // transpose block offset
    y = self.outer().index()[0] * TILE_DIM + self.inner().index()[1];

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
       odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
  }
};


// XXX need to figure out how to make this par(con) select grid_executor_2d automatically
auto grid(agency::size2 num_blocks, agency::size2 num_threads)
  -> decltype(agency::cuda::par(num_blocks, agency::cuda::con(num_threads)).on(agency::cuda::grid_executor_2d{}))
{
  return agency::cuda::par(num_blocks, agency::cuda::con(num_threads)).on(agency::cuda::grid_executor_2d{});
}


struct cuda_timer
{
  hipStream_t stream_;
  hipEvent_t start_;
  hipEvent_t end_;

  cuda_timer(hipStream_t stream = 0) : stream_(stream), start_{0}, end_{0}
  {
    hipEventCreate(&start_);
    hipEventCreate(&end_);
    reset();
  }

  void reset()
  {
    hipEventRecord(start_, stream_);
  }

  float elapsed_milliseconds() const
  {
    hipEventRecord(end_, stream_);
    hipEventSynchronize(end_);

    float result = 0;
    hipEventElapsedTime(&result, start_, end_);
    return result;
  }

  ~cuda_timer()
  {
    hipEventDestroy(start_);
    hipEventDestroy(end_);
  }
};


int main(int argc, char **argv)
{
  const int nx = 1024;
  const int ny = 1024;

  dim3 dimGrid(nx/TILE_DIM, ny/TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

  int devId = 0;
  if (argc > 1) devId = atoi(argv[1]);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, devId);
  printf("\nDevice : %s\n", prop.name);
  printf("Matrix size: %d %d, Block size: %d %d, Tile size: %d %d\n", 
         nx, ny, TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
  printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n",
         dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);
  
  hipSetDevice(devId);

  thrust::host_vector<float> h_idata(nx * ny);
  thrust::host_vector<float> h_cdata(nx * ny);
  thrust::host_vector<float> h_tdata(nx * ny);
  thrust::host_vector<float> gold(nx * ny);

  thrust::device_vector<float> d_idata(nx * ny);
  thrust::device_vector<float> d_cdata(nx * ny);
  thrust::device_vector<float> d_tdata(nx * ny);
  
  // check parameters and calculate execution configuration
  if (nx % TILE_DIM || ny % TILE_DIM) {
    throw std::logic_error("nx and ny must be a multiple of TILE_DIM");
  }

  if (TILE_DIM % BLOCK_ROWS) {
    throw std::logic_error("TILE_DIM must be a multiple of BLOCK_ROWS");
  }
    
  // initialize input
  std::iota(h_idata.begin(), h_idata.end(), 0);

  // correct result for error checking
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      gold[j*nx + i] = h_idata[i*nx + j];
  
  // copy input to device
  d_idata = h_idata;
  
  float ms;
  cuda_timer timer;

  // ------------
  // time kernels
  // ------------
  printf("%25s%25s\n", "Routine", "Bandwidth (GB/s)");
  
  // ----
  // copy 
  // ----
  printf("%25s", "copy");
  thrust::fill(d_cdata.begin(), d_cdata.end(), 0);
  // warm up
  agency::cuda::bulk_async(grid({dimGrid.x,dimGrid.y}, {dimBlock.x,dimBlock.y}), copy_kernel{}, raw_pointer_cast(d_cdata.data()), raw_pointer_cast(d_idata.data()));
  timer.reset();
  for(int i = 0; i < NUM_REPS; i++)
  {
    agency::cuda::bulk_async(grid({dimGrid.x,dimGrid.y}, {dimBlock.x,dimBlock.y}), copy_kernel{}, raw_pointer_cast(d_cdata.data()), raw_pointer_cast(d_idata.data()));
  }
  ms = timer.elapsed_milliseconds();
  h_cdata = d_cdata;
  postprocess(h_idata, h_cdata, ms);

  // -------------
  // copySharedMem 
  // -------------
  printf("%25s", "shared memory copy");
  thrust::fill(d_cdata.begin(), d_cdata.end(), 0);
  // warm up
  copySharedMem<<<dimGrid, dimBlock>>>(raw_pointer_cast(d_cdata.data()), raw_pointer_cast(d_idata.data()));
  timer.reset();
  for(int i = 0; i < NUM_REPS; i++)
  {
    copySharedMem<<<dimGrid, dimBlock>>>(raw_pointer_cast(d_cdata.data()), raw_pointer_cast(d_idata.data()));
  }
  ms = timer.elapsed_milliseconds();
  h_cdata = d_cdata;
  postprocess(h_idata, h_cdata, ms);

  // --------------
  // transposeNaive 
  // --------------
  printf("%25s", "naive transpose");
  thrust::fill(d_tdata.begin(), d_tdata.end(), 0);
  // warmup
  agency::cuda::bulk_async(grid({dimGrid.x,dimGrid.y}, {dimBlock.x,dimBlock.y}), transpose_naive{}, raw_pointer_cast(d_tdata.data()), raw_pointer_cast(d_idata.data()));
  timer.reset();
  for(int i = 0; i < NUM_REPS; i++)
  {
    agency::cuda::bulk_async(grid({dimGrid.x,dimGrid.y}, {dimBlock.x,dimBlock.y}), transpose_naive{}, raw_pointer_cast(d_tdata.data()), raw_pointer_cast(d_idata.data()));
  }
  ms = timer.elapsed_milliseconds();
  h_tdata = d_tdata;
  postprocess(gold, h_tdata, ms);

  // ------------------
  // transposeCoalesced 
  // ------------------
  printf("%25s", "coalesced transpose");
  thrust::fill(d_tdata.begin(), d_tdata.end(), 0);
  // warmup
  agency::cuda::bulk_async(grid({dimGrid.x,dimGrid.y}, {dimBlock.x,dimBlock.y}), transpose_coalesced{}, raw_pointer_cast(d_tdata.data()), raw_pointer_cast(d_idata.data()));
  timer.reset();
  for(int i = 0; i < NUM_REPS; i++)
  {
    agency::cuda::bulk_async(grid({dimGrid.x,dimGrid.y}, {dimBlock.x,dimBlock.y}), transpose_coalesced{}, raw_pointer_cast(d_tdata.data()), raw_pointer_cast(d_idata.data()));
  }
  ms = timer.elapsed_milliseconds();
  h_tdata = d_tdata;
  postprocess(gold, h_tdata, ms);

  // ------------------------
  // transposeNoBankConflicts
  // ------------------------
  printf("%25s", "conflict-free transpose");
  thrust::fill(d_tdata.begin(), d_tdata.end(), 0);
  // warmup
  agency::cuda::bulk_async(grid({dimGrid.x,dimGrid.y}, {dimBlock.x,dimBlock.y}), transpose_no_bank_conflicts{}, raw_pointer_cast(d_tdata.data()), raw_pointer_cast(d_idata.data()));
  timer.reset();
  for(int i = 0; i < NUM_REPS; i++)
  {
    agency::cuda::bulk_async(grid({dimGrid.x,dimGrid.y}, {dimBlock.x,dimBlock.y}), transpose_no_bank_conflicts{}, raw_pointer_cast(d_tdata.data()), raw_pointer_cast(d_idata.data()));
  }
  ms = timer.elapsed_milliseconds();
  h_tdata = d_tdata;
  postprocess(gold, h_tdata, ms);

  return 0;
}

